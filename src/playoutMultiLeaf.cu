
#include <hip/hip_runtime.h>
extern "C"
__global__ void playoutMultiLeaf(int *rands, int *numRands, int *board,
		int *boardWidth, int *colorToPlay, int *move, float *wins) {
	//copy board to local

	int l_width = *boardWidth;

	//NOTE!!!!! hardcoded!!! change if you can.....
	int tempBoard[9 * 9];
	for (int j = 0; j < l_width * l_width; j++) {
		tempBoard[j] = board[j];
	}

	int boardFull = 1;
	int wincolor = -1;
	int colorTP = *colorToPlay;

	tempBoard[move[blockIdx.x]] = colorTP;

	if (colorTP == 1){
		colorTP = 2;
	} else {
		colorTP = 1;
	}

	int count = 0;

	while (true) {
		count += 1;
		wincolor = -3;
		//actual playouts
		int n = rands[(blockIdx.x * (*numRands / blockDim.x) + count + threadIdx.x) % *numRands];
		if (tempBoard[n] == 0) {
			tempBoard[n] = colorTP;

			//check end of game.....

			int x = n % l_width;
			int y = n / l_width;
			int counter = 0;
			int xNew;
			int yNew;

			//DownR to UpL

			for (int j = -4; j < 4; j++) {
				xNew = x - j;
				yNew = y - j;
				if ((xNew >= 0) && (xNew < l_width) && (yNew >= 0)
						&& (yNew < l_width)) {
					if (tempBoard[xNew + yNew * l_width] == colorTP) {
						counter += 1;
					} else {
						counter = 0;
					}
					if (counter == 5) {
						break;
					}
					if ((j > 0) && (counter == 0)) {
						break;
					}
				}
			}
			if (counter == 5) {
				wincolor = colorTP;
				break;
			}

			counter = 0;

			//UpR to DownL

			for (int j = -4; j < 4; j++) {
				xNew = x - j;
				yNew = y + j;
				if ((xNew >= 0) && (xNew < l_width) && (yNew >= 0)
						&& (yNew < l_width)) {
					if (tempBoard[xNew + yNew * l_width] == colorTP) {
						counter += 1;
					} else {
						counter = 0;
					}
					if (counter == 5) {
						break;
					}
					if ((j > 0) && (counter == 0)) {
						break;
					}
				}
			}
			if (counter == 5) {
				wincolor = colorTP;
				break;
			}

			counter = 0;

			//horizontal check

			for (xNew = x - 4; xNew < x + 4; xNew++) {
				if ((xNew >= 0) && (xNew < l_width)) {
					if (tempBoard[xNew + y * l_width] == colorTP) {
						counter += 1;
					} else {
						counter = 0;
					}
					if (counter == 5) {
						break;
					}
					if ((xNew > x) && (counter == 0)) {
						break;
					}
				}
			}
			if (counter == 5) {
				wincolor = colorTP;
				break;
			}

			counter = 0;

			//vertical check

			for (yNew = y - 4; yNew < y + 4; yNew++) {
				if ((yNew >= 0) && (yNew < l_width)) {
					if (tempBoard[x + yNew * l_width] == colorTP) {
						counter += 1;
					} else {
						counter = 0;
					}
					if (counter == 5) {
						break;
					}
					if ((yNew > y) && (counter == 0)) {
						break;
					}
				}
			}
			if (counter == 5) {
				wincolor = colorTP;
				break;
			}

			//check if board is full
			for (int i = 0; i < (l_width * l_width); i++) {
				if (tempBoard[i] == 0) {
					boardFull = 0;
					break;
				}
			}
			if (boardFull) {
				wincolor = 0;
				break;
			}
			boardFull = 1;
			if (colorTP == 1) {
				colorTP = 2;
			} else {
				colorTP = 1;
			}
		}
	}
	if (wincolor == *colorToPlay) {
		atomicAdd(&wins[blockIdx.x], (float) 1.0);
	} else if (wincolor == 0) {
		atomicAdd(&wins[blockIdx.x], (float) 0.5);
	}
}
