
#include <hip/hip_runtime.h>
extern "C"
__global__ void playout(int *rands, int numRands, int *board, int const boardWidth, int colorToPlay, float *wins)
{
	//copy board to local

	//NOTE!!!!! hardcoded!!! change if you can.....
	int tempBoard[9*9];
	for (int j = 0; j < 9*9; j++){
		tempBoard[j] = board[j];
	}

	int boardFull = 1;
	int wincolor = -1;

	//actual playouts
	while (true){
		int n = rands[threadIdx.x % numRands];
		if (tempBoard[n] == 0){
			tempBoard[n] = colorToPlay;
			if (colorToPlay == 1){
				colorToPlay = 2;
			}
			else {
				colorToPlay = 1;
			}
		}

		//check end of game.....

		int color;
		int win = 0;

		int i;
		for (int x = 0; x < boardWidth; x++){
			for (int y = 0; y < boardWidth; y++){
				color = tempBoard[x + y*boardWidth];
				if (color != 0){
					win = 1;
					if ((x >= 4) && (y<= boardWidth -5)){
						for (i = 1; i < 5; i++){
							if (tempBoard[(x-i)+(y+i)*boardWidth] != color){
								win = 0;
								break;
							}
						}
						if (win){
							wincolor = color;
							break;
						}
					}
					win = 1;
					if (x <= boardWidth -5 && y <= boardWidth-5){
						for (i = 1; i < 5; i++){
							if (tempBoard[(x+i) + (y+i)*boardWidth] != color){
								win = 0;
								break;
							}
						}
						if (win){
							wincolor = color;
							break;
						}
					}
					win = 1;
					if (y <= boardWidth-5){
						for (i = 1; i < 5; i++){
							if (tempBoard[x + (y+i)*boardWidth] != color){
								win = 0;
								break;
							}
						}
						if (win){
							wincolor = color;
							break;
						}
					}
					win = 1;
					if (x <= boardWidth-5){
						for (i = 1; i < 5; i++){
							if (tempBoard[x+i+y*boardWidth] != color){
								win = false;
								break;
							}
						}
						if (win){
							wincolor = color;
							break;
						}
					}
				} else {
					boardFull = 0;
				}
			}
			if (wincolor != -1){
				break;
			}
		}
		if (wincolor != -1){
			break;
		}
	}
	if (boardFull) {
		atomicAdd(wins, (float)0.5);
	}
	else{
		if (wincolor != -1 && wincolor == colorToPlay){
			atomicAdd(wins, (float)1.0);
		}
	}
}
