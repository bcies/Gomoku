
#include <hip/hip_runtime.h>
extern "C"
__global__ void playout(int *rands, int *numRands, int *board, int *boardWidth, int *colorToPlay, float *wins)
{

	atomicAdd(wins, (float)1.0);

//	//copy board to local
//
//	//NOTE!!!!! hardcoded!!! change if you can.....
//	int tempBoard[9*9];
//	for (int j = 0; j < 9*9; j++){
//		tempBoard[j] = board[j];
//	}
//
//	int boardFull = 1;
//	int wincolor = -1;
//	int colorTP = *colorToPlay;
//
//	int count = 0;
//
//	//actual playouts
//	while (count < 1000){
//		int n = rands[threadIdx.x % *numRands];
//		if (tempBoard[n] == 0){
//			tempBoard[n] = colorTP;
//			if (colorTP == 1){
//				colorTP = 2;
//			}
//			else {
//				colorTP = 1;
//			}
//		}
//
//		//check end of game.....
//
//		int color;
//		int win = 0;
//
//		int i;
//		for (int x = 0; x < *boardWidth; x++){
//			for (int y = 0; y < *boardWidth; y++){
//				color = tempBoard[x + y * (*boardWidth)];
//				if (color != 0){
//					win = 1;
//					if ((x >= 4) && (y<= *boardWidth -5)){
//						for (i = 1; i < 5; i++){
//							if (tempBoard[(x-i)+(y+i) * (*boardWidth)] != color){
//								win = 0;
//
//								break;
//							}
//						}
//						if (win){
//							wincolor = color;
//							break;
//						}
//					}
//					win = 1;
//					if (x <= *boardWidth -5 && y <= *boardWidth-5){
//						for (i = 1; i < 5; i++){
//							if (tempBoard[(x+i) + (y+i) * (*boardWidth)] != color){
//								win = 0;
//								break;
//							}
//						}
//						if (win){
//							wincolor = color;
//							break;
//						}
//					}
//					win = 1;
//					if (y <= *boardWidth-5){
//						for (i = 1; i < 5; i++){
//							if (tempBoard[x + (y+i) * (*boardWidth)] != color){
//								win = 0;
//								break;
//							}
//						}
//						if (win){
//							wincolor = color;
//							break;
//						}
//					}
//					win = 1;
//					if (x <= *boardWidth-5){
//						for (i = 1; i < 5; i++){
//							if (tempBoard[x+i+y * (*boardWidth)] != color){
//								win = false;
//								break;
//							}
//						}
//						if (win){
//							wincolor = color;
//							break;
//						}
//					}
//				} else {
//					boardFull = 0;
//				}
//			}
//			if (wincolor != -1){
//				break;
//			}
//		}
//		if (wincolor != -1){
//			break;
//		}
//		count++;
//	}
//	if (boardFull) {
//		atomicAdd(wins, (float)0.5);
//	}
//	else{
//		if (wincolor != -1 && wincolor == colorTP){
//			atomicAdd(wins, (float)1.0);
//		}
//	}
}
